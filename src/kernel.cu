#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

glm::vec3 *dev_SortedPos;
glm::vec3 *dev_sortedVel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.

  //DONE
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int)); 
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");
  
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");


  //2.3
  hipMalloc((void**)&dev_SortedPos, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_sortedVel, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO <<<fullBlocksPerGrid, blockSize >>>(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO <<<fullBlocksPerGrid, blockSize >>>(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/



/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/

//DONE

__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids

    
    glm::vec3 perceivedCenter(0); //For Rule 1
    glm::vec3 c(0); //For Rule 2
    glm::vec3 perceivedVelocity(0); //For Rule 3;
    
    float rule1Boids = 0, rule3Boids = 0;

    for (int i = 0; i < N; i++) {
        if (i != iSelf) {

            float distance = glm::distance(pos[i], pos[iSelf]);

            if (distance < rule1Distance) {
                perceivedCenter += pos[i];
                rule1Boids++;
            }

            if (distance < rule2Distance) {
                c -= pos[i] - pos[iSelf];
            }

            if (distance < rule3Distance) {
                perceivedVelocity += vel[i];
                rule3Boids++;
            }
        }
    }

    if (rule1Boids > 0) {
        perceivedCenter = perceivedCenter / rule1Boids;    //Rule 1
    }

    //No modifications for Rule 2

    if (rule3Boids > 0) {
        perceivedVelocity = perceivedVelocity / rule3Boids;    //Rule 3
    }
    
    return ((perceivedCenter - pos[iSelf]) * rule1Scale) + (c * rule2Scale) + (perceivedVelocity * rule3Scale);
}

/**
* TODO-1.2 implement basic flocking  
* For each of the `N` bodies, update its position based on its current velocity.
*/

//DONE

__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3* pos,
    glm::vec3* vel1, glm::vec3* vel2) {
    // Compute a new velocity based on pos and vel1
    // Clamp the speed
    // Record the new velocity into vel2. Question: why NOT vel1?
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    glm::vec3 tempVel = vel1[index] + computeVelocityChange(N, index, pos, vel1);
    
    if (glm::length(tempVel) > 1.0f) {

        tempVel = glm::normalize(tempVel);
    }

    vel2[index] = tempVel;


}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2

    //DONE
    int iSelf = threadIdx.x + (blockIdx.x * blockDim.x);
    if (iSelf >= N) {
        return;
    }

    glm::vec3 curr = pos[iSelf];
    glm::vec3 cell = floor((curr - gridMin) * inverseCellWidth);
    int cellIdx = gridIndex3Dto1D(cell.x, cell.y, cell.z, gridResolution);
    gridIndices[iSelf] = cellIdx;

    indices[iSelf] = iSelf;
    gridIndices[iSelf] = cellIdx;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

//2.3
__global__ void kernSortPosVel(int N, int* particleArrayIndices, glm::vec3* pos, glm::vec3* vel, glm::vec3* sortedPos, glm::vec3* sortedVel) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    sortedPos[index] = pos[particleArrayIndices[index]];
    sortedVel[index] = vel[particleArrayIndices[index]];
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

    //DONE

    /*
    int start = particleGridIndices[0];
    gridCellStartIndices[start] = 0;

    for (int i = 1; i < N; i++) {
        if (particleGridIndices[i] != start) {
            gridCellEndIndices[start] = i - 1;
            start = particleGridIndices[i];
            gridCellStartIndices[start] = i;

            if (i == N - 1) {
                gridCellEndIndices[start] == i;
            }
        }      
    }
*/
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }

    int start = -1;
    int end = -1;
    int gridIdx = particleGridIndices[index];

    for (int i = 0; i < N; i++) {
        if (particleGridIndices[i] == gridIdx) {
            start = i;
            while (i < N && particleGridIndices[i] == gridIdx) {
                i++;
            }
            end = i - 1;
            break;
        }
    }
    gridCellStartIndices[gridIdx] = start;
    gridCellEndIndices[gridIdx] = end;
}

//2.1 helper function
__device__ glm::vec3 computeVel(int N, int cellIdx, int iSelf, int gridResolution, int* gridCellStartIndices,
    int* gridCellEndIndices, int* particleArrayIndices, glm::vec3* pos, glm::vec3* vel) {

    if (cellIdx < 0 || cellIdx >= gridResolution * gridResolution * gridResolution) {
        return glm::vec3(0);
    }

    
    int start = gridCellStartIndices[cellIdx];
    int end = gridCellEndIndices[cellIdx];

    glm::vec3 currPos = pos[iSelf];

    float rule1Boids = 0, rule3Boids = 0;

    glm::vec3 perceivedCenter(0); //For Rule 1
    glm::vec3 c(0); //For Rule 2
    glm::vec3 perceivedVelocity(0); //For Rule 3;


    for (int i = start; i <= end; i++) {

        int arrayIdx = particleArrayIndices[i];

        if (arrayIdx == iSelf) {
            continue;
        }

        float distance = glm::distance(currPos, pos[arrayIdx]);

        if (distance < rule1Distance) {
            perceivedCenter += pos[arrayIdx];
            rule1Boids++;
        }

        if (distance < rule2Distance) {
            c -= pos[arrayIdx] - currPos;
        }

        if (distance < rule3Distance) {
            perceivedVelocity += vel[arrayIdx];
            rule3Boids++;
        }

    }

    if (rule1Boids > 0) {
        perceivedCenter = perceivedCenter / rule1Boids;    //Rule 1
    }

    //No modifications for Rule 2

    if (rule3Boids > 0) {
        perceivedVelocity = perceivedVelocity / rule3Boids;    //Rule 3
    }

    return ((perceivedCenter - currPos) * rule1Scale) + 
                    (c * rule2Scale) + (perceivedVelocity * rule3Scale);
    

    
}

//2.1 helper function
__device__ glm::vec3 velUpdateHelper(int N, int iSelf, int gridResolution, glm::vec3 gridMin,
    float inverseCellWidth, float cellWidth,
    int* gridCellStartIndices, int* gridCellEndIndices,
    int* particleArrayIndices,
    glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {

    glm::vec3 curr = pos[iSelf];
    glm::vec3 gridCell = floor((curr - gridMin) * inverseCellWidth);
    glm::vec3 rounded = round((curr - gridMin) * inverseCellWidth);
    //int cellIdx = gridIndex3Dto1D(gridCell[0], gridCell[1], gridCell[2], gridResolution);

    glm::vec3 gMin(0);
    glm::vec3 gMax(0);

    //X
    if (gridCell.x == rounded.x) {
        gMin.x = gridCell.x - 1;
        gMax.x = gridCell.x;
    }
    else {
        gMin.x = gridCell.x;
        gMax.x = gridCell.x + 1;
    }

    //Y
    if (gridCell.y == rounded.y) {
        gMin.y = gridCell.y - 1;
        gMax.y = gridCell.y;
    }
    else {
        gMin.y = gridCell.y;
        gMax.y = gridCell.y + 1;
    }

    //Z
    if (gridCell.z == rounded.z) {
        gMin.z = gridCell.z - 1;
        gMax.z = gridCell.z;
    }
    else {
        gMin.z = gridCell.z;
        gMax.z = gridCell.z + 1;
    }

    glm::vec3 tempVel(0);

    for (int k = gMin.z; k <= gMax.z; k++) {
        for (int j = gMin.y; j <= gMax.y; j++) {
            for (int i = gMin.x; i <= gMax.x; i++) {
                int cellIdx = gridIndex3Dto1D(i, j, k, gridResolution);
                tempVel += computeVel(N, cellIdx, iSelf, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
            }
        }
    }
    
    return tempVel;

}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2


    //DONE

    int iSelf = threadIdx.x + (blockIdx.x * blockDim.x);
    if (iSelf > N - 1) {
        return;
    }

    glm::vec3 tempVel = vel1[iSelf] + velUpdateHelper(N, iSelf, gridResolution, gridMin, inverseCellWidth, cellWidth,
        gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2);

    if (glm::length(tempVel) > maxSpeed) {
        tempVel = glm::normalize(tempVel);

        vel2[iSelf] = tempVel;
    }

}

//2.3helper function
__device__ glm::vec3 computeVel_Coherent(int N, int gridResolution, int cellIdx, int iSelf,
    int* gridCellStartIndices, int* gridCellEndIndices, glm::vec3* pos, glm::vec3* vel) {

    if (cellIdx < 0 || cellIdx >= gridResolution * gridResolution * gridResolution) {
        return glm::vec3(0);
    }

   

    int start = gridCellStartIndices[cellIdx];
    int end = gridCellEndIndices[cellIdx];
 
    if (start < 0) {
        return glm::vec3(0);
    }

    glm::vec3 currPos = pos[iSelf];

    float rule1Boids = 0, rule3Boids = 0;

    glm::vec3 perceivedCenter(0); //For Rule 1
    glm::vec3 c(0); //For Rule 2
    glm::vec3 perceivedVelocity(0); //For Rule 3;


    for (int i = start; i <= end; i++) {

        if (i == iSelf) {
            continue;
        }

        

        float distance = glm::distance(currPos, pos[i]);

        if (distance < rule1Distance) {
            perceivedCenter += pos[i];
            rule1Boids++;
        }

        if (distance < rule2Distance) {
            c -= pos[i] - currPos;
        }

        if (distance < rule3Distance) {
            perceivedVelocity += vel[i];
            rule3Boids++;
        }

    }

    if (rule1Boids > 0) {
        perceivedCenter = perceivedCenter / rule1Boids;    //Rule 1
    }

    //No modifications for Rule 2

    if (rule3Boids > 0) {
        perceivedVelocity = perceivedVelocity / rule3Boids;    //Rule 3
    }

    return ((perceivedCenter - currPos) * rule1Scale) +
        (c * rule2Scale) + (perceivedVelocity * rule3Scale);


}

//2.3 helper function
__device__ glm::vec3 velUpdateHelper_Coherent(int N, int iSelf, int gridResolution, glm::vec3 gridMin,
    float inverseCellWidth, float cellWidth, int* gridCellStartIndices, int* gridCellEndIndices,
    glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {

    glm::vec3 curr = pos[iSelf];
    glm::vec3 gridCell = floor((curr - gridMin) * inverseCellWidth);
    glm::vec3 rounded = round((curr - gridMin) * inverseCellWidth);

    glm::vec3 gMin(0);
    glm::vec3 gMax(0);

    //X
    if (gridCell.x == rounded.x) {
        gMin.x = gridCell.x - 1;
        gMax.x = gridCell.x;
    }
    else {
        gMin.x = gridCell.x;
        gMax.x = gridCell.x + 1;
    }

    //Y
    if (gridCell.y == rounded.y) {
        gMin.y = gridCell.y - 1;
        gMax.y = gridCell.y;
    }
    else {
        gMin.y = gridCell.y;
        gMax.y = gridCell.y + 1;
    }

    //Z
    if (gridCell.z == rounded.z) {
        gMin.z = gridCell.z - 1;
        gMax.z = gridCell.z;
    }
    else {
        gMin.z = gridCell.z;
        gMax.z = gridCell.z + 1;
    }

     glm::vec3 tempVel(0);

    for (int k = gMin.z; k <= gMax.z; k++) {
        for (int j = gMin.y; j <= gMax.y; j++) {
            for (int i = gMin.x; i <= gMax.x; i++) {
                int cellIdx = gridIndex3Dto1D(i, j, k, gridResolution);
                tempVel += computeVel_Coherent(N, gridResolution, cellIdx, iSelf, gridCellStartIndices, gridCellEndIndices, pos, vel1);
            }
        }
    }
    
    return tempVel;

}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }

    glm::vec3 tempVel(0);

    tempVel = vel1[index] + velUpdateHelper_Coherent(N, index, gridResolution, gridMin, inverseCellWidth, cellWidth,
        gridCellStartIndices, gridCellEndIndices, pos, vel1, vel2);

    if (glm::length(tempVel) > maxSpeed) {
        tempVel = glm::normalize(tempVel);
    }

    vel2[index] = tempVel;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

    kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize >>>(numObjects, dev_pos, dev_vel1, dev_vel2);
    kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (numObjects, dt, dev_pos, dev_vel2);

  // TODO-1.2 ping-pong the velocity buffers
    dev_vel1 = dev_vel2;

}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // -> label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // ~ Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // -> Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // ~ Perform velocity updates using neighbor search
  // -> Update positions
  // -> Ping-pong buffers as needed


    //DONE
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount,
        gridMinimum, gridInverseCellWidth,
        dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

    //SORT
    thrust::device_ptr<int> dev_thrust_particleArrayIndices(dev_particleArrayIndices);
    thrust::device_ptr<int> dev_thrust_particleGridIndices(dev_particleGridIndices);
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);


    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices,
        dev_gridCellStartIndices, dev_gridCellEndIndices);

    kernUpdateVelNeighborSearchScattered <<<fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount, 
        gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, 
        dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel1);

    dev_vel1 = dev_vel2;
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

    thrust::device_ptr<int> dev_thrust_particleArrayIndices(dev_particleArrayIndices);
    thrust::device_ptr<int> dev_thrust_particleGridIndices(dev_particleGridIndices);
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    kernSortPosVel << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_pos, dev_vel1, dev_SortedPos, dev_sortedVel);

    kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
        dev_SortedPos, dev_sortedVel, dev_vel2);

    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_SortedPos, dev_sortedVel);
    dev_vel1 = dev_vel2;
    dev_pos = dev_SortedPos;
}


void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.

  //DONE
  hipFree(dev_particleGridIndices);
  hipFree(dev_particleArrayIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  //2.3
  hipFree(dev_SortedPos);
  hipFree(dev_sortedVel);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
