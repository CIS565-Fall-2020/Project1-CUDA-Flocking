#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  glm::vec3 velChange1(0.0f, 0.0f, 0.0f);
  glm::vec3 perceivedCenter(0.0f, 0.0f, 0.0f);
  int numNeighbors1 = 0;
  for (int i = 0; i < N; i++) {
    if (i != iSelf && glm::distance(pos[i], pos[iSelf]) < rule1Distance) {
      perceivedCenter += pos[i];
      numNeighbors1++;
    }
  }
  if (numNeighbors1 > 0)
  {
    perceivedCenter /= numNeighbors1;
    velChange1 = (perceivedCenter - pos[iSelf]) * rule1Scale;
  }

  // Rule 2: boids try to stay a distance d away from each other
  glm::vec3 velChange2(0.0f, 0.0f, 0.0f);
  for (int i = 0; i < N; i++) {
    if (i != iSelf && glm::distance(pos[i], pos[iSelf]) < rule2Distance) {
      velChange2 -= (pos[i] - pos[iSelf]);
    }
  }
  velChange2 *= rule2Scale;

  // Rule 3: boids try to match the speed of surrounding boids
  glm::vec3 velChange3(0.0f, 0.0f, 0.0f);
  glm::vec3 perceivedVel(0.0f, 0.0f, 0.0f);
  int numNeighbors3 = 0;
  for (int i = 0; i < N; i++) {
    if (i != iSelf && glm::distance(pos[i], pos[iSelf]) < rule3Distance) {
      perceivedVel += vel[i];
      numNeighbors3++;
    }
  }
  if (numNeighbors3 > 0)
  {
    perceivedVel /= numNeighbors3;
    velChange3 = perceivedVel * rule3Scale;
  }
  return velChange1 + velChange2 + velChange3;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  // Compute a new velocity based on pos and vel1
  glm::vec3 newVel = vel1[index] + computeVelocityChange(N, index, pos, vel1);

  // Clamp the speed
  float speed = glm::length(newVel);
  if (speed > maxSpeed) {
    newVel = glm::normalize(newVel);
  }

  // Record the new velocity into vel2. Question: why NOT vel1?
  vel2[index] = newVel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1 - done
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    int boidIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (boidIndex >= N) {
      return;
    }
    glm::vec3 boidGridCell = glm::floor(inverseCellWidth * (pos[boidIndex] - gridMin));
    gridIndices[boidIndex] = gridIndex3Dto1D(boidGridCell.x, boidGridCell.y, boidGridCell.z, gridResolution);
    indices[boidIndex] = boidIndex;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1 - done
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
  int index = (blockIdx.x * blockDim.x) + threadIdx.x; // grid index
  if (index >= N) {
    return;
  }
  
  for (int i = 0; i < N; i++) {
    if (particleGridIndices[i] == index) {
      // Find start
      if (i == 0 || particleGridIndices[i - 1] != particleGridIndices[i]) {
        gridCellStartIndices[index] = i;
      }
      // Find end
      if (i == N - 1 || particleGridIndices[i + 1] != particleGridIndices[i]) {
        gridCellEndIndices[index] = i;
        break;
      }
    }
  }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - done 
  // Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  int boidIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (boidIndex >= N)
  {
    return;
  }
  glm::vec3 boidGridCell = inverseCellWidth * (pos[boidIndex] - gridMin); // might not be rounded/aka integer
  glm::vec3 boidGridCellFloor = glm::floor(boidGridCell); // must be rounded/aka integer
  // The two below has each component within [-1, 1] as integers. Basically tells 
  // the range relative to the current boid cell that neighbors might be in
  int3 minCell = make_int3(0, 0, 0);
  int3 maxCell = make_int3(0, 0, 0);
  if (boidGridCellFloor.x > 0 && glm::fract(boidGridCell.x) < 0.5f) minCell.x = -1;
  if (boidGridCellFloor.y > 0 && glm::fract(boidGridCell.y) < 0.5f) minCell.y = -1;
  if (boidGridCellFloor.z > 0 && glm::fract(boidGridCell.z) < 0.5f) minCell.z = -1;
  if (boidGridCellFloor.x < gridResolution - 1 && glm::fract(boidGridCell.x) > 0.5f) maxCell.x = 1;
  if (boidGridCellFloor.y < gridResolution - 1 && glm::fract(boidGridCell.y) > 0.5f) maxCell.y = 1;
  if (boidGridCellFloor.z < gridResolution - 1 && glm::fract(boidGridCell.z) > 0.5f) maxCell.z = 1 ;
  
  // velocity change due to each rule
  glm::vec3 velChange1(0.0f, 0.0f, 0.0f); // boids try to fly towards center of mass of neighboring boids
  glm::vec3 velChange2(0.0f, 0.0f, 0.0f); // boids try to keep a small distance away from other objects/boid
  glm::vec3 velChange3(0.0f, 0.0f, 0.0f); // boids try to match velocity with nearby boids
  int numNeighbors1 = 0;
  int numNeighbors3 = 0;
  
  for (int i = minCell.x + boidGridCellFloor.x; i < maxCell.x + boidGridCellFloor.x; i++) {
    for (int j = minCell.y + boidGridCellFloor.y; j < maxCell.y + boidGridCellFloor.y; j++) {
      for (int k = minCell.z + boidGridCellFloor.z; k < maxCell.z + boidGridCellFloor.z; k++) {
        int curCellIndex = gridIndex3Dto1D(i, j, k, gridResolution);
        if (gridCellStartIndices[curCellIndex] > -1) {
          for (int b = gridCellStartIndices[curCellIndex]; b <= gridCellEndIndices[curCellIndex]; b++) {
            int boidBIndex = particleArrayIndices[b]; // index of pos, vel1 or vel2 of a neighbor boid
            if (boidBIndex != boidIndex) {
              float dist = glm::distance(pos[boidBIndex], pos[boidIndex]);
              if (dist < rule1Distance) {
                // Technically, we are finding the perceived center for this point (will be averaged out later, then scale to turn into velocity)
                velChange1 += pos[boidBIndex];
                numNeighbors1++;
              }
              if (dist < rule2Distance) velChange2 -= (pos[boidBIndex] - pos[boidIndex]);
              if (dist < rule3Distance) {
                // Technically, we are finding the perceived velocity for this point (will be averaged out later, then scale to turn into correct velocity)
                velChange3 += vel1[boidBIndex];
                numNeighbors3++;
              }
            }
          }
        }
      }
    }
  }

  // Finalize velocity change from rule 1, 2 and 3
  if (numNeighbors1 > 0) {
    velChange1 /= numNeighbors1;
    velChange1 = (velChange1  - pos[boidIndex]) * rule1Scale;
  }
  velChange2 *= rule2Scale;
  if (numNeighbors3 > 0) {
    velChange3 /= numNeighbors3;
    velChange3 *= rule3Scale;
  }
  vel2[boidIndex] = vel1[boidIndex] + velChange1 + velChange2 + velChange3;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);

  // TODO-1.2 ping-pong the velocity buffers
  glm::vec3* temp = dev_vel1;
  dev_vel1 = dev_vel2;
  dev_vel2 = temp;
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1 - done
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum,
    gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(gridCellCount, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  
  kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
    dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
  
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
  kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);

  glm::vec3* temp = dev_vel1;
  dev_vel1 = dev_vel2;
  dev_vel2 = temp;
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 - done - TODO-2.3 - Free any additional buffers here.
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");

  // test kernComputeIndices -------------------
 /* N = 5;
  int* dev_testPos;
  int* dev_testGridCellStartIndices;
  int* dev_testGridCellEndIndices;
  int* dev_testParticleArrayIndices;
  int* dev_particleGridIndices;
  
  std::unique_ptr<glm::vec3[]>testPos{ new glm::vec3[N] };
  testPos[0] = glm::vec3(0.f, 0.f, 0.f);
  testPos[1] = glm::vec3(1.f, 1.f, 1.f);
  testPos[2] = glm::vec3(3.f, 3.f, 3.f);
  testPos[3] = glm::vec3(5.f, 5.f, 5.f);
  testPos[4] = glm::vec3(6.f, 6.f, 6.f);
  hipMemcpy(dev_testPos, testPos.get(), sizeof(glm::vec3) * N, hipMemcpyHostToDevice);*/

  
  // test kernIdentifyCellStartEnd -------------
  return;
}
