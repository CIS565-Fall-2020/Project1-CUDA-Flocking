#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 1024

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_particlePosReshuffle;
glm::vec3 *dev_particleVelReshuffle;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_pointer_cast<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_pointer_cast<int>(dev_particleGridIndices);

  hipMalloc((void**)&dev_particlePosReshuffle, N * sizeof(glm::vec3));
  hipMalloc((void**)&dev_particleVelReshuffle, N * sizeof(glm::vec3));

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
    glm::vec3 newVelocity = vel[iSelf];

    float3 center = make_float3(0.0f, 0.0f, 0.0f);
    float3 separate = make_float3(0.0f, 0.0f, 0.0f);
    float3 cohesion = make_float3(0.0f, 0.0f, 0.0f);

    int neighborCount1 = 0;
    int neighborCount3 = 0;

    float3 thisBoidPos = make_float3(pos[iSelf].x, pos[iSelf].y, pos[iSelf].z);

  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    for (int i = 0; i < N; i++) {
        if (i == iSelf) continue;
        float distance = glm::length(pos[i] - pos[iSelf]);
        float3 thatBoidPos = make_float3(pos[i].x, pos[i].y, pos[i].z);
        if (distance < rule1Distance) {
            center.x += thatBoidPos.x;
            center.y += thatBoidPos.y;
            center.z += thatBoidPos.z;
            neighborCount1++;
        }

        // Rule 2: boids try to stay a distance d away from each other
        if (distance < rule2Distance) {
            separate.x -= thatBoidPos.x - thisBoidPos.x;
            separate.y -= thatBoidPos.y - thisBoidPos.y;
            separate.z -= thatBoidPos.z - thisBoidPos.z;
        }
        // Rule 3: boids try to match the speed of surrounding boids
        if(distance < rule3Distance) {
            cohesion.x += vel[i].x;
            cohesion.y += vel[i].y;
            cohesion.z += vel[i].z;
            neighborCount3++;
        }
    }

    glm::vec3 v1(0.0f, 0.0f, 0.0f);
    glm::vec3 v2 = v1;
    glm::vec3 v3 = v1;
    if (neighborCount1 > 0) {
        center.x /= neighborCount1;
        center.y /= neighborCount1;
        center.z /= neighborCount1;

        v1.x = (center.x - thisBoidPos.x) * rule1Scale;
        v1.y = (center.y - thisBoidPos.y) * rule1Scale;
        v1.z = (center.z - thisBoidPos.z) * rule1Scale;

    }
    if (neighborCount3 > 0) {
        v3.x = cohesion.x * rule3Scale;
        v3.y = cohesion.y * rule3Scale;
        v3.z = cohesion.z * rule3Scale;
        v3 /= neighborCount3;

    }

    v2.x = separate.x * rule2Scale;
    v2.y = separate.y * rule2Scale;
    v2.z = separate.z * rule2Scale;

    return v1 + v2 + v3 + vel[iSelf];
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    glm::vec3 newVelocity = computeVelocityChange(N, index, pos, vel1);
  // Clamp the speed
    float speed = glm::length(newVelocity);
    if (speed > maxSpeed) {
        newVelocity = (newVelocity / speed) * maxSpeed;
    }
  // Record the new velocity into vel2. Question: why NOT vel1?
    vel2[index] = newVelocity;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }
    glm::vec3 thisBoidPos = pos[index];
    glm::vec3 gridPos = glm::floor((thisBoidPos - gridMin) * inverseCellWidth);
    int gridIndex = gridIndex3Dto1D((int)gridPos.x, (int)gridPos.y, (int)gridPos.z, gridResolution);
    gridIndices[index] = gridIndex;

    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }
    int curGridIndex = particleGridIndices[index];
    if (index == 0) {
        gridCellStartIndices[curGridIndex] = index;
        return;
    }
    if (index == N - 1) {
        gridCellEndIndices[curGridIndex] = index;
    }
    int prevGridIndex = particleGridIndices[index - 1];


    // If the current grid index not equals to the previous one, 
    // update both the start index of the current grid index 
    // and the end index of the previous grid index
    if (curGridIndex != prevGridIndex) {
        gridCellStartIndices[curGridIndex] = index;
        gridCellEndIndices[prevGridIndex] = index - 1;
    }

    return;    
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }
    glm::vec3 thisBoidPos = pos[index];
    glm::vec3 gridPos = glm::floor((thisBoidPos - gridMin) * inverseCellWidth);
    int gridIndex = gridIndex3Dto1D((int)gridPos.x, (int)gridPos.y, (int)gridPos.z, gridResolution);

  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
    int neighborCount1 = 0;
    int neighborCount3 = 0;

    glm::vec3 center(0.0f, 0.0f, 0.0f);
    glm::vec3 separate(0.0f, 0.0f, 0.0f);
    glm::vec3 cohesion(0.0f, 0.0f, 0.0f);
    glm::vec3 thisBoidNewVel = vel1[index];

    for (int k = -1; k <= 1; k++) {
        for (int j = -1; j <= 1; j++) {
            for (int i = -1; i <= 1; i++) { 
                int neighGridX = (int) gridPos.x + i;
                int neighGridY = (int) gridPos.y + j;
                int neighGridZ = (int)gridPos.z + k;

                int neighGridIndex = gridIndex3Dto1D(neighGridX, neighGridY, neighGridZ, gridResolution);
                int startIndex = gridCellStartIndices[neighGridIndex];
                int endIndex = gridCellEndIndices[neighGridIndex];

                if (startIndex < 0 || startIndex >= N || endIndex < 0 || endIndex >= N) continue;

                for (int idx = startIndex; idx <= endIndex; idx++) {
                    int boidIndex = particleArrayIndices[idx];
                    if (boidIndex == index) continue;

                    glm::vec3 thatBoidPos = pos[boidIndex];
                    glm::vec3 thatBoidVel = vel1[boidIndex];
                    float distance = glm::length(thisBoidPos - thatBoidPos);

                    if (distance < rule1Distance) {
                        center += thatBoidPos;
                        neighborCount1++;
                    }

                    if (distance < rule2Distance) {
                        separate -= (thatBoidPos - thisBoidPos);
                    }

                    if (distance < rule3Distance) {
                        cohesion += thatBoidVel;
                        neighborCount3++;
                    }
                }

            }
        }
    }
    if (neighborCount1 > 0) {
        center /= neighborCount1;
        thisBoidNewVel += (center - thisBoidPos) * rule1Scale;

    }

    if (neighborCount3 > 0) {
        thisBoidNewVel += cohesion * rule3Scale;
    }

    thisBoidNewVel += separate * rule2Scale;

    float speed = glm::length(thisBoidNewVel);
    if (speed > maxSpeed) {
        thisBoidNewVel = thisBoidNewVel * maxSpeed / speed;
    }

    vel2[index] = thisBoidNewVel;
}

__global__ void kernReshuffle(int N, int* indices, glm::vec3* attribute, glm::vec3* shuffleArrtribute) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    shuffleArrtribute[index] = attribute[indices[index]];
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }
    glm::vec3 thisBoidPos = pos[index];
    glm::vec3 gridPos = glm::floor((thisBoidPos - gridMin) * inverseCellWidth);
    int gridIndex = gridIndex3Dto1D((int)gridPos.x, (int)gridPos.y, (int)gridPos.z, gridResolution);

  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.

  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
    int neighborCount1 = 0;
    int neighborCount3 = 0;

    glm::vec3 center(0.0f, 0.0f, 0.0f);
    glm::vec3 separate(0.0f, 0.0f, 0.0f);
    glm::vec3 cohesion(0.0f, 0.0f, 0.0f);
    glm::vec3 thisBoidNewVel = vel1[index];

    for (int k = -1; k <= 1; k++) {
        for (int j = -1; j <= 1; j++) {
            for (int i = -1; i <= 1; i++) {
                int neighGridX = (int)gridPos.x + i;
                int neighGridY = (int)gridPos.y + j;
                int neighGridZ = (int)gridPos.z + k;

                int neighGridIndex = gridIndex3Dto1D(neighGridX, neighGridY, neighGridZ, gridResolution);
                int startIndex = gridCellStartIndices[neighGridIndex];
                int endIndex = gridCellEndIndices[neighGridIndex];

                if (startIndex < 0 || startIndex >= N || endIndex < 0 || endIndex >= N) continue;

                for (int idx = startIndex; idx <= endIndex; idx++) {

                    glm::vec3 thatBoidPos = pos[idx];
                    if (thatBoidPos == thisBoidPos) continue;
                    glm::vec3 thatBoidVel = vel1[idx];
                    float distance = glm::length(thisBoidPos - thatBoidPos);

                    if (distance < rule1Distance) {
                        center += thatBoidPos;
                        neighborCount1++;
                    }

                    if (distance < rule2Distance) {
                        separate -= (thatBoidPos - thisBoidPos);
                    }

                    if (distance < rule3Distance) {
                        cohesion += thatBoidVel;
                        neighborCount3++;
                    }
                }

            }
        }
    }
    if (neighborCount1 > 0) {
        center /= neighborCount1;
        thisBoidNewVel += (center - thisBoidPos) * rule1Scale;
    }

    if (neighborCount3 > 0) {
        thisBoidNewVel += cohesion * rule3Scale;
    }

    thisBoidNewVel += separate * rule2Scale;

    float speed = glm::length(thisBoidNewVel);
    if (speed > maxSpeed) {
        thisBoidNewVel = thisBoidNewVel * maxSpeed / speed;
    }

    vel2[index] = thisBoidNewVel;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernUpdateVelocityBruteForce <<<fullBlocksPerGrid, threadsPerBlock >>> (numObjects, dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");
    // TODO-1.2 ping-pong the velocity buffers
    hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernComputeIndices <<<fullBlocksPerGrid, threadsPerBlock>>> (
                                                numObjects, 
                                                gridSideCount, 
                                                gridMinimum, 
                                                gridInverseCellWidth, 
                                                dev_pos, 
                                                dev_particleArrayIndices, 
                                                dev_particleGridIndices);
    checkCUDAErrorWithLine(" kernComputeIndices failed!");

  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
      // Wrap device vectors in thrust iterators for use with thrust.

    // LOOK-2.1 Example for using thrust::sort_by_key
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("failed at kernResetIntBuffer");
    kernResetIntBuffer << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_gridCellEndIndices, -1);
    checkCUDAErrorWithLine("failed at kernResetIntBuffer");
    kernIdentifyCellStartEnd <<<fullBlocksPerGrid, threadsPerBlock>>> (
                                                    numObjects, 
                                                    dev_particleGridIndices, 
                                                    dev_gridCellStartIndices, 
                                                    dev_gridCellEndIndices);

  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered <<<fullBlocksPerGrid, threadsPerBlock >>> (
                                                                    numObjects, 
                                                                    gridSideCount,
                                                                    gridMinimum, 
                                                                    gridInverseCellWidth, 
                                                                    gridCellWidth, 
                                                                    dev_gridCellStartIndices, 
                                                                    dev_gridCellEndIndices,
                                                                    dev_particleArrayIndices,
                                                                    dev_pos,
                                                                    dev_vel1,
                                                                    dev_vel2);

  // - Update positions
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");
  // - Ping-pong buffers as needed
    hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernComputeIndices << <fullBlocksPerGrid, threadsPerBlock >> > (
                                                    numObjects,
                                                    gridSideCount,
                                                    gridMinimum,
                                                    gridInverseCellWidth,
                                                    dev_pos,
                                                    dev_particleArrayIndices,
                                                    dev_particleGridIndices);
    checkCUDAErrorWithLine(" kernComputeIndices failed!");

  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("failed at kernResetIntBuffer");
    kernResetIntBuffer << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_gridCellEndIndices, -1);
    checkCUDAErrorWithLine("failed at kernResetIntBuffer");
    kernIdentifyCellStartEnd << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects,
        dev_particleGridIndices,
        dev_gridCellStartIndices,
        dev_gridCellEndIndices);

  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
    kernReshuffle << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_particleArrayIndices, dev_pos, dev_particlePosReshuffle);
    checkCUDAErrorWithLine("failed at kernReshuffle Pos");
    kernReshuffle << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_particleArrayIndices, dev_vel1, dev_particleVelReshuffle);
    checkCUDAErrorWithLine("failed at kernReshuffle Vel");

  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, threadsPerBlock >> > (
                                                                        numObjects,
                                                                        gridSideCount,
                                                                        gridMinimum,
                                                                        gridInverseCellWidth,
                                                                        gridCellWidth,
                                                                        dev_gridCellStartIndices,
                                                                        dev_gridCellEndIndices,
                                                                        dev_particlePosReshuffle,
                                                                        dev_particleVelReshuffle,
                                                                        dev_vel2);
  // - Update positions
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_particlePosReshuffle, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");
  // - Ping-pong buffers as needed
    hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_pos, dev_particlePosReshuffle, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_particlePosReshuffle);
  hipFree(dev_particleVelReshuffle);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
